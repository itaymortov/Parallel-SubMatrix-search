#include "hip/hip_runtime.h"

#include "CalcMatch.h"
#include <math.h>
#include "func.h"


__global__ void calcMatchingGPU(int* pic, int picSize, int* obj, int objSize, double matchingValue, int* matchCount, int* matchPlace, int objId, int* hasMatch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i <= picSize - objSize && j <= picSize - objSize)
    {
        double match = 0;
        for (int x = 0; x < objSize; x++)
        {
            for (int y = 0; y < objSize; y++)
            {
                double p = pic[(i+x)*picSize + j+y];
                double o = obj[x*objSize + y];
                double matchAdd = (p - o);
                if(matchAdd<0)
                    matchAdd = matchAdd * - 1;
                matchAdd = matchAdd/p;
                match = match + matchAdd;
            }
        }
        match = match / (objSize * objSize);
        if (match <= matchingValue && *matchCount < 3 && *hasMatch != 1)
        {
        	atomicAdd(hasMatch, 1);
            
            matchPlace[*matchCount*3] = objId;
            matchPlace[*matchCount*3+1] = i;
            matchPlace[*matchCount*3+2] = j;
        }
    }
}

void calcMatchingGPU(struct Picture* pic, struct Objects* obj, double matchingValue, int* matchCount,int* match,int* matchPlace)
{
    int* d_Pic;
    int* d_Obj;
    int* d_MatchCount;
    int* d_MatchPlace;
    int* d_hasMatch; 
	
    // Allocate memory on the device
    hipMalloc(&d_Pic, pic->picSize * pic->picSize * sizeof(int));
    hipMalloc(&d_Obj, obj->objSize * obj->objSize * sizeof(int));
    hipMalloc(&d_MatchCount, sizeof(int));
    hipMalloc(&d_MatchPlace, 3 * sizeof(int));
    hipMalloc(&d_hasMatch, sizeof(int));
	
    // Copy data from host to device
    hipMemcpy(d_Pic, pic->pic, pic->picSize * pic->picSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Obj, obj->obj, obj->objSize * obj->objSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_MatchCount, matchCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemset(d_hasMatch, 0, sizeof(int));
	hipMemset(d_MatchPlace, 0, 3 * sizeof(int));
	
		
    // Launch the kernel
    dim3 block(32, 32);
    //grid makes it so their is enough threads to get from the beginning of the matrix to the end minus the object size, that so we will not go over the matrix size.
    //in the calcMatchingGPU function we have a for the size of the object size so that we can calc the border of the pictures (from picSize-objSize to picSize)
    dim3 grid((pic->picSize - obj->objSize + block.x - 1) / block.x, (pic->picSize - obj->objSize + block.y - 1) / block.y);
    
    
    calcMatchingGPU<<<grid, block>>>(d_Pic, pic->picSize, d_Obj, obj->objSize, matchingValue, d_MatchCount, d_MatchPlace, obj->objId ,d_hasMatch);
	
    // Copy data from device to host
    hipMemcpy(matchCount, d_MatchCount, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(match, d_hasMatch, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(matchPlace, d_MatchPlace, 3 * sizeof(int), hipMemcpyDeviceToHost);	
    //only if we have a match we are adding 1 to matchCount.
    if(*match == 1){
        *matchCount++;
    }

    // Free memory on the device
    hipFree(d_Pic);
    hipFree(d_Obj);
    hipFree(d_MatchPlace);
    hipFree(d_MatchCount);
    hipFree(d_hasMatch);

}
